#include "hip/hip_runtime.h"
#include <iostream>
#include "priority_queue.cuh"
#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime.h>
#include <hip/device_functions.h>
#include <hip/hip_runtime_api.h>

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <iostream>

#include <fstream>
#include <string>

using namespace std;


#define NUM_NODES 5

typedef struct
{
    int start;     // Index of first adjacent node in Ea    
    int length;    // Number of adjacent nodes 
} Node2;


__global__ void CUDA_A_STAR_KERNEL(Node2 *Va, int *Ea, bool *Fa, bool *Xa, int *Ca,bool *done)
{

    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id > NUM_NODES)
        *done = false;


    if (Fa[id] == true && Xa[id] == false)
    {
        printf("%d ", id); //This printf gives the order of vertices in BFS 
        Fa[id] = false;
        Xa[id] = true;
        __syncthreads(); 
        //int k = 0;
        //int i;
        int start = Va[id].start;
        int end = start + Va[id].length;
        for (int i = start; i < end; i++) 
        {
            int nid = Ea[i];


            //printf(" nid %d\n", nid );
            //cout<<nid<<endl;
            
            if (Xa[nid] == false)
            {
                Ca[nid] = Ca[id] + 1;
                Fa[nid] = true;
                *done = false;
            }

        }

    }

}

const int tam_map = 100;

char m[tam_map][tam_map];

void cargar_mapa(){

    //CARGANDO EL MAPA
  
    //int w, h;

      string line;
      ifstream myfile ("mapa.csv");

      //w = h = tam_map;  
       //vector<vector<char>> mapa(100,vector<char>(100,0));
      int x=0;
      int y=0;
      int value=0;

      if (myfile.is_open())
      {
        while ( getline (myfile,line) )
        {
          if(x<=tam_map-1){

            y=0;
            for (int i = 0; i < line.size(); ++i)
            {
                if(line[i]=='1' || line[i]=='0' ){
                  
                  //mapa[x][y] = line[i];

                  value = (int)line[i] - 48;
                  m[x][y] = value;
                  //cout<<m[x][y]<<" ";
                  y++;  
                }
                
            }
            //cout<<endl;
            //cout <<'\n';
            x++;
          }

        }
        myfile.close();
      }

      else cout << "Unable to open file";

}


__global__ void print_mapGPU(char *map)
{
    // Thread indexing within Grid - note these are
    // in column major order.
    //int tidx = threadIdx.x + blockIdx.x * blockDim.x;
    //int tidy = threadIdx.y + blockIdx.y * blockDim.y;

    // a_ij = a[i][j], where a is in row major order
    //int a_ij = a[tidy +  tidx*N];
    for (int i = 0; i < 20; ++i)
    {
        for (int j = 0; j < 20; ++j)
        {
            printf("%d ", map[i*j]);
        }
        printf("\n");
    }

} 



void print_mapCPU()
{
    for (int i = 0; i < 20; ++i)
    {
        for (int j = 0; j < 20; ++j)
        {
            printf("%d ", m[i][j] );
        }
        printf("\n");
    }
} 


int main(){

    //CARGANDO EL MAPA GPU
    cargar_mapa();


    //Copiar el mapa al device

    char *map;
    //char *map_d;
    const size_t map_size = sizeof(char) * size_t(tam_map*tam_map);
    hipMalloc((void **)&map, map_size); 
    
    hipMemcpy(map, m, map_size, hipMemcpyHostToDevice);

    cout<<"MAP EN CPU: "<<endl;
    print_mapCPU();

    cout<<"MAP EN GPU: "<<endl;
    print_mapGPU<<<1,1>>>(map); 
    hipDeviceSynchronize();


    //CARGANDO LA PRIORITY QUEUE EN EL KERNEL
    hipError_t err = hipSuccess;
    printf("CUDA A START WORKING... \n");
    
    //int num_threads = STR_LENGTH;

    int num_threads = 1;
    int num_blocks = 1;

    int tam = 5;
    //size_t size = tam*sizeof(int);

    size_t size = tam*sizeof(Node);

    //int *nodes_host = malloc(tam * sizeof(int));


    /*
    int* nodes_host = (int*)malloc(size);
    int* nodes_device = (int*)malloc(size);
    */

    Node* nodes_host = (Node*)malloc(size);
    Node* nodes_device = (Node*)malloc(size);

    /*
    Point* start = (Point) malloc(sizeof(*Point));
    start.x = 0; start.y=0;

    //start = {0,0};

    //Point end = (Point)malloc(sizeof(Point));
    //start.x = 100; start.y = 100;

    Point *end = (end*) malloc(sizeof(*end));
    end  = {100,100};
    */


    //size_t tam_node = sizeof(Point);

    //hipMalloc(&start_d, tam_node);
    //hipMalloc(&end_d, tam_node);

    hipMalloc(&nodes_device, size);
    queue<<<num_blocks,num_threads>>>(nodes_device,15);

    //point& s, point& e, map& mp
    //search<<<num_blocks,num_threads>>>(start,end,nodes_device);

    hipDeviceSynchronize();

    err = hipMemcpy(nodes_host,nodes_device,size,hipMemcpyDeviceToHost);

    hipFree(nodes_device);

    if (err != hipSuccess)
    {
        fprintf(stderr, "error code %s)!\n", hipGetErrorString(err));
        //exit(EXIT_FAILURE);
    }

    
    for(int i = 0; i < tam; i++)
	{
	  	printf(" costo:%d distancia:%d posicion:%d,%d  parent:%d,%d  \n", nodes_host[i].cost, nodes_host[i].dist, nodes_host[i].pos.x,nodes_host[i].pos.y , nodes_host[i].parent.x, nodes_host[i].parent.y );
	    //printf(" distancia:%d \n", *(nodes_host[i])->dist);   
    }


}